/**********************************************************
"Hello World"-type program to test different srun layouts.

Written by Tom Papatheodore
**********************************************************/

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <iomanip>
#include <iomanip>
#include <string.h>
#include <mpi.h>
#include <sched.h>
#include <omp.h>

// Macro for checking errors in GPU API calls
#define gpuErrorCheck(call)                                                                  \
do{                                                                                          \
    hipError_t gpuErr = call;                                                               \
    if(hipSuccess != gpuErr){                                                               \
        printf("GPU Error - %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(gpuErr)); \
        exit(0);                                                                             \
    }                                                                                        \
}while(0)

int getgpu(const int rank, const int section, const char * name) {
    const char* gpu_id_list;

    // If CUDA_VISIBLE_DEVICES is set, capture visible GPUs
    const char* gpu_visible_devices = getenv("CUDA_VISIBLE_DEVICES");
    if(gpu_visible_devices == NULL){
       	gpu_id_list = "N/A";
    }
    else{
       	gpu_id_list = gpu_visible_devices;
    }

	// Find how many GPUs runtime says are available
	int num_devices = 0;
    gpuErrorCheck( hipGetDeviceCount(&num_devices) );

	if (num_devices > 0){
		char busid[64];
        std::string busid_list = "";
        std::string rt_gpu_id_list = "";

		// Loop over the GPUs available to each MPI rank
		for(int i=0; i<num_devices; i++){

			gpuErrorCheck( hipSetDevice(i) );

			// Get the PCIBusId for each GPU and use it to query for UUID
			gpuErrorCheck( hipDeviceGetPCIBusId(busid, 64, i) );

			// Concatenate per-MPIrank GPU info into strings for print
            if(i > 0) rt_gpu_id_list.append(",");
            rt_gpu_id_list.append(std::to_string(i));

            std::string temp_busid(busid);

            if(i > 0) busid_list.append(",");
//            busid_list.append(temp_busid.substr(8,2));
            busid_list.append(temp_busid);

            printf("MPI %03d - SEC %d - Node %s - RT_GPU_ID %s - GPU_ID %s - Bus_ID %s\n",
                rank, section, name, rt_gpu_id_list.c_str(), gpu_id_list, busid_list.c_str());
		}
	}
	return 0;
}
